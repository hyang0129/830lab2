
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<string.h>
#include<algorithm>
#include<queue>

using namespace std;

int V,D,E,L,K,A,B,C,M,Q;
int* X;
int* edges;

int squared_l2_dist(int* x,int* y,int D){
	int sum2 = 0;
	for(int i = 0;i < D;++i)
		sum2 += (x[i] - y[i]) * (x[i] - y[i]);
	return sum2;
}


//__global__ void cuda_squared_l2_dist(int* origin, int* nodes, int* distances) {
//
//	int index = threadIdx.x + blockDim.x * blockIdx.x;
//	int* x = nodes[index];
//
//	int sum2 = 0;
//	for (int i = 0; i < D; ++i)
//		sum2 += (origin[i] - x[i]) * (origin[i] - x[i]);
//
//	distances[index] = sum2;
//}


vector<int> explore(int start_point, int max_hop) {
	queue<pair<int, int>> q;
	vector<int> nodes;
	q.push(make_pair(start_point, 0));
	nodes.push_back(start_point);
	while (!q.empty()) {
		auto now = q.front();
		q.pop();
		int id = now.first;
		int hop = now.second;

		if (hop + 1 <= max_hop) {
			int degree = edges[id * (L + 1)];

			for (int i = 1; i <= degree; ++i) {
				int v = edges[id * (L + 1) + i];

				q.push(std::make_pair(v, hop + 1));
				nodes.push_back(v);
			}

		}
	}

	return nodes;
}

int main(int argc,char** argv){
	FILE* fin = fopen(argv[1],"r");
	FILE* fout = fopen(argv[2],"w");
	fscanf(fin,"%d%d%d%d%d%d%d%d%d%d",&V,&D,&E,&L,&K,&A,&B,&C,&M,&Q);
	X = new int[V * D];
	for(int i = 0;i < K;++i)
		fscanf(fin,"%d",&X[i]);
	for(int i = K;i < V * D;++i)
		X[i] = ((long long)A * X[i - 1] + (long long)B * X[i - 2] + C) % M;
	edges = new int[V * (L + 1)];
	for(int i = 0;i < V;++i){
		edges[i * (L + 1)] = 0;
	}
	for(int i = 0;i < E;++i){
		int u,v;
		fscanf(fin,"%d%d",&u,&v);
		int degree = edges[u * (L + 1)];
		edges[u * (L + 1) + degree + 1] = v;
		++edges[u * (L + 1)];
	}


	int* query_data = new int[D];

	// cuda 
	//cudaMallocManaged(&query_data, D * sizeof(int));
	
	for(int i = 0;i < Q;++i){
		int start_point,hop;
		fscanf(fin,"%d%d",&start_point,&hop);
		for(int i = 0;i < D;++i){
			fscanf(fin,"%d",&query_data[i]);
		}

		// explore for all nodes 
		vector<int> allPossibleNodes = explore(start_point, hop);

		

		// non cuda component 
		// 
		int* distances = new int[allPossibleNodes.size()];

		int* targets = new int[allPossibleNodes.size()*D];
		
		//targets = new int*[allPossibleNodes.size()];
		//for (int j = 0; j < allPossibleNodes.size(); ++j) {
		//	targets[j] = new int[1]; 
		//}


		// cuda 
		//cudaMallocManaged(&targets, D * allPossibleNodes.size() * sizeof(int));
		//cudaMallocManaged(&distances, allPossibleNodes.size() * sizeof(int));
		// cuda 


		for (int j = 0; j < allPossibleNodes.size(); ++j) {
			int p = j * D;

			int* temp = X + allPossibleNodes.at(j) * D;

			for (int k = 0; k < D; k++) {
				targets[p + k] = temp[k];
			}
		}



		// non cuda
		
		for (int j = 0; j < allPossibleNodes.size(); ++j) {
			distances[j] = squared_l2_dist(targets + j * D, query_data, D);
		}

		// non cuda 
		

		//cuda 

		//int threadsPerBlock = 256;
		//int blocksPerGrid = (allPossibleNodes.size() + threadsPerBlock - 1) / threadsPerBlock;

		//cuda_squared_l2_dist << <blocksPerGrid, threadsPerBlock > >> (query_data, targets, distances);
		//cudaDeviceSynchronize();
		// 
		// 
		//cuda 

		// get min 
		int min_d = 2147483647;
		int min_id = 2147483647;
		for (int j = 0; j < allPossibleNodes.size(); ++j) {
			int id = allPossibleNodes.at(j);
			int d = distances[j];

			if (d < min_d || (d == min_d && id < min_id)) {
				min_d = d;
				min_id = id;
			}
		}

		fprintf(fout,"%d\n",min_id);
	}
	fclose(fin);
	fclose(fout);

	delete[] X;
	delete[] edges;
	delete[] query_data;

	return 0;
}

