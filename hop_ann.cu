#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>
#include<algorithm>
#include<queue>
#include<iostream>

int V, D, E, L, K, A, B, C, M, Q;
int* X_d;
int* X;
int* edges;

int squared_l2_dist(int* x, int* y, int D) {
	int sum2 = 0;
	for (int i = 0; i < D; ++i)
		sum2 += (x[i] - y[i]) * (x[i] - y[i]);
	return sum2;
}

__global__ void computeParallel(int* X, int* query, int D, int* hop, int* id, int* d) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int sum2 = 0;
	int* x = X + id[index] * D;

	int* y = query;
	for (int i = 0; i < D; ++i) {
		sum2 += (x[i] - y[i]) * (x[i] - y[i]);
	}

	d[index] = sum2;
}

int nearest_id(int start_point, int max_hop, int* query_data, int* X) {

	//std::cout<<"Edges : ";
	//for (int i = 0; i<V*(L+1); ++i)
	//	std::cout<<edges[i];
	//std::cout<<"\n";

	std::queue<std::pair<int, int>> q;
	q.push(std::make_pair(start_point, 0));
	int min_d = std::numeric_limits<int>::max();
	int min_id = -1;
	do {
		int* id;
		int* hop;
		int siz = q.size();
		//std::cout<<"Size : "<<siz<<"\n";
		hipMallocManaged(&id, siz * sizeof(int));
		hipMallocManaged(&hop, siz * sizeof(int));
		int ctr = 0;
		while (!q.empty()) {
			auto now = q.front();
			q.pop();
			id[ctr] = now.first;
			hop[ctr] = now.second;

			//std::cout<<"ID : "<<id[ctr]<<" "<<"HOP : "<<hop[ctr]<<"\n";
			ctr++;
		}
		int* d_d;
		hipMallocManaged(&d_d, siz * sizeof(int));
		computeParallel << <siz, 1 >> > (X, query_data, D, hop, id, d_d);
		//hipDeviceSynchronize();

		int* d = new int[siz];
		hipMemcpy(d, d_d, siz * sizeof(int), hipMemcpyDeviceToHost);

		for (int i = 0; i < siz; ++i) {
			//std::cout<<"Node rn : "<<id[i]<<" " <<"Hop : "<<hop[i]<<"\n";
			if ((d[i] < min_d) || (d[i] == min_d && id[i] < min_id)) {
				min_d = d[i];
				min_id = id[i];
			}
			if (hop[i] + 1 <= max_hop) {
				//std::cout<<"Nodes being inserted for id : "<<id[i]<< " : ";
				int degree = edges[id[i] * (L + 1)];
				//std::cout<<"Degree : "<<degree<<" : ";
				for (int j = 1; j <= degree; ++j) {
					int v = edges[id[i] * (L + 1) + j];
					//std::cout<<"ID : "<< id[i] << " L+1 : "<<L+1<<" Inter : "<<id[i]*(L+1)<<" Index : "<<id[i] * (L + 1) + i<<" Value : "<<v<<" ";
					q.push(std::make_pair(v, hop[i] + 1));
				}
				//std::cout<<"\n";
			}
		}
		//std::cout<<"\n";
		hipFree(id);
		hipFree(hop);
		hipFree(d);
	} while (!q.empty());
	//std::cout<<"\n\n";
	return min_id;
}

int main(int argc, char** argv) {
	FILE* fin = fopen(argv[1], "r");
	FILE* fout = fopen(argv[2], "w");
	fscanf(fin, "%d%d%d%d%d%d%d%d%d%d", &V, &D, &E, &L, &K, &A, &B, &C, &M, &Q);
	X = new int[V * D];



	for (int i = 0; i < K; ++i)
		fscanf(fin, "%d", &X[i]);

	for (int i = K; i < V * D; ++i)
	{
		X[i] = ((long long)A * X[i - 1] + (long long)B * X[i - 2] + C) % M;
	}


	edges = new int[V * (L + 1)];
	for (int i = 0; i < V; ++i) {
		edges[i * (L + 1)] = 0;
	}
	for (int i = 0; i < E; ++i) {
		int u, v;
		fscanf(fin, "%d%d", &u, &v);
		int degree = edges[u * (L + 1)];
		edges[u * (L + 1) + degree + 1] = v;
		++edges[u * (L + 1)];
	}
	std::cout << "Edges: ";

	for (int i = 0; i < V * (L + 1); ++i) {
		std::cout << edges[i] << " ";
	}
	hipMallocManaged(&X_d, (V * D) * sizeof(int));
	hipMemcpy(X_d, X, V * D * sizeof(int), hipMemcpyHostToDevice);

	//int* query_data = new int[D];
	//int* query_data_d;
	//hipMallocManaged(&query_data_d, D * sizeof(int));
	//for (int i = 0; i < Q; ++i) {
	//	int start_point, hop;
	//	fscanf(fin, "%d%d", &start_point, &hop);
	//	for (int i = 0; i < D; ++i) {
	//		fscanf(fin, "%d", &query_data[i]);
	//	}
	//	hipMemcpy(query_data_d, query_data, D * sizeof(int), hipMemcpyHostToDevice);
	//	fprintf(fout, "%d\n", nearest_id(start_point, hop, query_data_d, X_d));
	//}


	int* query_data = new int[D];

	hipMallocManaged(&query_data, D * sizeof(int));
	for (int i = 0; i < Q; ++i) {
		int start_point, hop;
		fscanf(fin, "%d%d", &start_point, &hop);
		for (int i = 0; i < D; ++i) {
			fscanf(fin, "%d", &query_data[i]);
		}
		fprintf(fout, "%d\n", nearest_id(start_point, hop, query_data_d, X_d));
	}


	fclose(fin);
	fclose(fout);
	//delete[] X;
	delete[] edges;
	//delete[] query_data;
	hipFree(X);
	hipFree(query_data);
	return 0;
}

